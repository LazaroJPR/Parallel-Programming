#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16
#define SIZE 1000

__global__ void gpu_matrix_mult(int *a, int *b, int *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (col < k && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}

__global__ void gpu_matrix_mult_transpose(int *a, int *bT, int *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (col < m && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * bT[col * n + i];
        }
        c[row * m + col] = sum;
    }
}

void cpu_matrix_mult(int **h_a, int **h_b, int **h_c, int m, int n, int k) {
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            int tmp = 0;
            for (int h = 0; h < n; ++h)
            {
                tmp += h_a[i][h] * h_b[h][j];
            }
            h_c[i][j] = tmp;
        }
    }
}

void cpu_matrix_mult_transpose(int **h_a, int **h_bT, int **h_c, int m, int n) {
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < m; ++j)
        {
            int tmp = 0;
            for (int h = 0; h < n; ++h)
            {
                tmp += h_a[i][h] * h_bT[j][h];
            }
            h_c[i][j] = tmp;
        }
    }
}

void transpose_matrix(int **B, int **B_T, int n, int k) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < k; j++) {
            B_T[j][i] = B[i][j];  // Transpor a matriz B
        }
    }
}

int main(int argc, char const *argv[])
{
    int m = SIZE, n = SIZE, k = SIZE;
    srand(3333);

    int **h_a = (int **)malloc(SIZE * sizeof(int *));
    int **h_b = (int **)malloc(SIZE * sizeof(int *));
    int **h_bT = (int **)malloc(SIZE * sizeof(int *));
    int **h_c = (int **)malloc(SIZE * sizeof(int *));
    int **h_cc = (int **)malloc(SIZE * sizeof(int *));
    for (int i = 0; i < SIZE; i++) {
        h_a[i] = (int *)malloc(SIZE * sizeof(int));
        h_b[i] = (int *)malloc(SIZE * sizeof(int));
        h_bT[i] = (int *)malloc(SIZE * sizeof(int));
        h_c[i] = (int *)malloc(SIZE * sizeof(int));
        h_cc[i] = (int *)malloc(SIZE * sizeof(int));
    }

    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            h_a[i][j] = rand() % 1024;
            h_b[i][j] = rand() % 1024;
        }
    }

    transpose_matrix(h_b, h_bT, n, k);

    clock_t start_cpu_normal = clock();
    cpu_matrix_mult(h_a, h_b, h_cc, m, n, k);
    clock_t end_cpu_normal = clock();
    double cpu_time_normal = ((double)(end_cpu_normal - start_cpu_normal)) / CLOCKS_PER_SEC;

    clock_t start_cpu_transpose = clock();
    cpu_matrix_mult_transpose(h_a, h_bT, h_c, m, n);
    clock_t end_cpu_transpose = clock();
    double cpu_time_transpose = ((double)(end_cpu_transpose - start_cpu_transpose)) / CLOCKS_PER_SEC;

    int *d_a, *d_b, *d_bT, *d_c;
    hipMalloc((void **) &d_a, sizeof(int) * m * n);
    hipMalloc((void **) &d_b, sizeof(int) * n * k);
    hipMalloc((void **) &d_bT, sizeof(int) * n * m);  // B^T agora é n * m
    hipMalloc((void **) &d_c, sizeof(int) * m * k);

    int *h_a_linear = (int *)malloc(m * n * sizeof(int));
    int *h_b_linear = (int *)malloc(n * k * sizeof(int));
    int *h_bT_linear = (int *)malloc(n * m * sizeof(int));
    int *h_c_linear = (int *)malloc(m * k * sizeof(int));

    for (int i = 0; i < m; ++i)
        for (int j = 0; j < n; ++j)
            h_a_linear[i * n + j] = h_a[i][j];

    for (int i = 0; i < n; ++i)
        for (int j = 0; j < k; ++j)
            h_b_linear[i * k + j] = h_b[i][j];

    for (int i = 0; i < n; ++i)
        for (int j = 0; j < m; ++j)
            h_bT_linear[j * n + i] = h_bT[j][i];

    hipMemcpy(d_a, h_a_linear, sizeof(int) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b_linear, sizeof(int) * n * k, hipMemcpyHostToDevice);
    hipMemcpy(d_bT, h_bT_linear, sizeof(int) * n * m, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    hipEventRecord(start);
    gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds_normal = 0;
    hipEventElapsedTime(&milliseconds_normal, start, stop);

    hipEventRecord(start);
    gpu_matrix_mult_transpose<<<dimGrid, dimBlock>>>(d_a, d_bT, d_c, m, n, k);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds_transpose = 0;
    hipEventElapsedTime(&milliseconds_transpose, start, stop);

    printf("Multiplicar as linhas da matriz A com a matriz B e salvar em C (CPU)\n");
    printf("Tempo de execucao: %f seconds\n", cpu_time_normal);
    printf("Multiplicar as linhas da matriz A com as linhas da matriz transposta B e salvar em C (CPU)\n");
    printf("Tempo de execucao: %f seconds\n", cpu_time_transpose);
    printf("Multiplicar as linhas da matriz A com a matriz B e salvar em C (GPU)\n");
    printf("Tempo de execucao: %f milliseconds\n", milliseconds_normal);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_bT);
    hipFree(d_c);

    for (int i = 0; i < SIZE; i++) {
        free(h_a[i]);
        free(h_b[i]);
        free(h_bT[i]);
        free(h_c[i]);
        free(h_cc[i]);
    }
    free(h_a);
    free(h_b);
    free(h_bT);
    free(h_c);
    free(h_cc);

    free(h_a_linear);
    free(h_b_linear);
    free(h_bT_linear);
    free(h_c_linear);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}